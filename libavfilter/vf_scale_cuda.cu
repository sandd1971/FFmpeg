#include "hip/hip_runtime.h"
/*
 * This file is part of FFmpeg.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cuda/vector_helpers.cuh"
#include "vf_scale_cuda.h"

typedef float4 (*coeffs_function_t)(float, float);

__device__ inline float4 lanczos_coeffs(float x, float param)
{
    const float pi = 3.141592654f;

    float4 res = make_float4(
        pi * (x + 1),
        pi * x,
        pi * (x - 1),
        pi * (x - 2));

    res.x = res.x == 0.0f ? 1.0f :
        __sinf(res.x) * __sinf(res.x / 2.0f) / (res.x * res.x / 2.0f);
    res.y = res.y == 0.0f ? 1.0f :
        __sinf(res.y) * __sinf(res.y / 2.0f) / (res.y * res.y / 2.0f);
    res.z = res.z == 0.0f ? 1.0f :
        __sinf(res.z) * __sinf(res.z / 2.0f) / (res.z * res.z / 2.0f);
    res.w = res.w == 0.0f ? 1.0f :
        __sinf(res.w) * __sinf(res.w / 2.0f) / (res.w * res.w / 2.0f);

    return res / (res.x + res.y + res.z + res.w);
}

__device__ inline float4 bicubic_coeffs(float x, float param)
{
    const float A = param == SCALE_CUDA_PARAM_DEFAULT ? 0.0f : -param;

    float4 res;
    res.x = ((A * (x + 1) - 5 * A) * (x + 1) + 8 * A) * (x + 1) - 4 * A;
    res.y = ((A + 2) * x - (A + 3)) * x * x + 1;
    res.z = ((A + 2) * (1 - x) - (A + 3)) * (1 - x) * (1 - x) + 1;
    res.w = 1.0f - res.x - res.y - res.z;

    return res;
}

template<typename V>
__device__ inline V apply_coeffs(float4 coeffs, V c0, V c1, V c2, V c3)
{
    V res = c0 * coeffs.x;
    res  += c1 * coeffs.y;
    res  += c2 * coeffs.z;
    res  += c3 * coeffs.w;

    return res;
}

template<typename T>
__device__ inline void Subsample_Nearest(hipTextureObject_t tex,
                                         T *dst,
                                         int dst_width, int dst_height, int dst_pitch,
                                         int src_width, int src_height,
                                         int bit_depth)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;

        dst[yo*dst_pitch+xo] = tex2D<T>(tex, xi, yi);
    }
}

template<typename T>
__device__ inline void Subsample_Bilinear(hipTextureObject_t tex,
                                          T *dst,
                                          int dst_width, int dst_height, int dst_pitch,
                                          int src_width, int src_height,
                                          int bit_depth)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        // 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
        float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        // Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);

        intT r = { 0 };
        vec_set_scalar(r, 2);
        r += tex2D<T>(tex, xi - dx, yi - dy);
        r += tex2D<T>(tex, xi + dx, yi - dy);
        r += tex2D<T>(tex, xi - dx, yi + dy);
        r += tex2D<T>(tex, xi + dx, yi + dy);
        vec_set(dst[yo*dst_pitch+xo], r >> 2);
    }
}

template<typename T>
__device__ inline void Subsample_Bicubic(coeffs_function_t coeffs_function,
                                         hipTextureObject_t tex,
                                         T *dst,
                                         int dst_width, int dst_height, int dst_pitch,
                                         int src_width, int src_height,
                                         int bit_depth, float param)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale - 0.5f;
        float yi = (yo + 0.5f) * vscale - 0.5f;
        float px = floor(xi);
        float py = floor(yi);
        float fx = xi - px;
        float fy = yi - py;

        float factor = bit_depth > 8 ? 0xFFFF : 0xFF;

        float4 coeffsX = coeffs_function(fx, param);
        float4 coeffsY = coeffs_function(fy, param);

#define PIX(x, y) tex2D<floatT>(tex, (x), (y))

        dst[yo * dst_pitch + xo] = from_floatN<T, floatT>(
            apply_coeffs<floatT>(coeffsY,
                apply_coeffs<floatT>(coeffsX, PIX(px - 1, py - 1), PIX(px, py - 1), PIX(px + 1, py - 1), PIX(px + 2, py - 1)),
                apply_coeffs<floatT>(coeffsX, PIX(px - 1, py    ), PIX(px, py    ), PIX(px + 1, py    ), PIX(px + 2, py    )),
                apply_coeffs<floatT>(coeffsX, PIX(px - 1, py + 1), PIX(px, py + 1), PIX(px + 1, py + 1), PIX(px + 2, py + 1)),
                apply_coeffs<floatT>(coeffsX, PIX(px - 1, py + 2), PIX(px, py + 2), PIX(px + 1, py + 2), PIX(px + 2, py + 2))
            ) * factor
        );

#undef PIX
    }
}

extern "C" {

#define NEAREST_KERNEL(T) \
    __global__ void Subsample_Nearest_ ## T(hipTextureObject_t src_tex,                  \
                                            T *dst,                                       \
                                            int dst_width, int dst_height, int dst_pitch, \
                                            int src_width, int src_height,                \
                                            int bit_depth)                                \
    {                                                                                     \
        Subsample_Nearest<T>(src_tex, dst,                                                \
                              dst_width, dst_height, dst_pitch,                           \
                              src_width, src_height,                                      \
                              bit_depth);                                                 \
    }

NEAREST_KERNEL(uchar)
NEAREST_KERNEL(uchar2)
NEAREST_KERNEL(uchar4)

NEAREST_KERNEL(ushort)
NEAREST_KERNEL(ushort2)
NEAREST_KERNEL(ushort4)

#define BILINEAR_KERNEL(T) \
    __global__ void Subsample_Bilinear_ ## T(hipTextureObject_t src_tex,                  \
                                             T *dst,                                       \
                                             int dst_width, int dst_height, int dst_pitch, \
                                             int src_width, int src_height,                \
                                             int bit_depth)                                \
    {                                                                                      \
        Subsample_Bilinear<T>(src_tex, dst,                                                \
                              dst_width, dst_height, dst_pitch,                            \
                              src_width, src_height,                                       \
                              bit_depth);                                                  \
    }

BILINEAR_KERNEL(uchar)
BILINEAR_KERNEL(uchar2)
BILINEAR_KERNEL(uchar4)

BILINEAR_KERNEL(ushort)
BILINEAR_KERNEL(ushort2)
BILINEAR_KERNEL(ushort4)

#define BICUBIC_KERNEL(T) \
    __global__ void Subsample_Bicubic_ ## T(hipTextureObject_t src_tex,                  \
                                            T *dst,                                       \
                                            int dst_width, int dst_height, int dst_pitch, \
                                            int src_width, int src_height,                \
                                            int bit_depth, float param)                   \
    {                                                                                     \
        Subsample_Bicubic<T>(&bicubic_coeffs, src_tex, dst,                               \
                             dst_width, dst_height, dst_pitch,                            \
                             src_width, src_height,                                       \
                             bit_depth, param);                                           \
    }

BICUBIC_KERNEL(uchar)
BICUBIC_KERNEL(uchar2)
BICUBIC_KERNEL(uchar4)

BICUBIC_KERNEL(ushort)
BICUBIC_KERNEL(ushort2)
BICUBIC_KERNEL(ushort4)


#define LANCZOS_KERNEL(T) \
    __global__ void Subsample_Lanczos_ ## T(hipTextureObject_t src_tex,                  \
                                            T *dst,                                       \
                                            int dst_width, int dst_height, int dst_pitch, \
                                            int src_width, int src_height,                \
                                            int bit_depth, float param)                   \
    {                                                                                     \
        Subsample_Bicubic<T>(&lanczos_coeffs, src_tex, dst,                               \
                             dst_width, dst_height, dst_pitch,                            \
                             src_width, src_height,                                       \
                             bit_depth, param);                                           \
    }

LANCZOS_KERNEL(uchar)
LANCZOS_KERNEL(uchar2)
LANCZOS_KERNEL(uchar4)

LANCZOS_KERNEL(ushort)
LANCZOS_KERNEL(ushort2)
LANCZOS_KERNEL(ushort4)

}
